#include "hip/hip_runtime.h"
﻿#include <opencv2/core.hpp>
#include <opencv2/imgproc.hpp>
#include <opencv2/highgui.hpp>

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include <iostream>

using namespace std;
using namespace cv;

// Ядро CUDA для вычисления LBP
__global__ void calculate_lbp(const uchar* input_image, int* output_codes, int* histogram, int width_image, int height_image, int radius_neighbors, int count_neighbors)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x; // Координата x текущего пикселя
    int y = blockIdx.y * blockDim.y + threadIdx.y; // Координата y текущего пикселя

    if (x >= width_image || y >= height_image)
        return;

    double const M_PI = 3.1415926535f;

    int center = y * width_image + x;

    unsigned char code = 0;
    for (int n = 0; n < count_neighbors; n++)
    {
        double angle = 2 * M_PI * n / count_neighbors;
        int neighbor_x = x + radius_neighbors * cos(angle);
        int neighbor_y = y + radius_neighbors * sin(angle);

        if (neighbor_x >= 0 && neighbor_x < width_image && neighbor_y >= 0 && neighbor_y < height_image)
        {
            if (input_image[neighbor_y * width_image + neighbor_x] >= input_image[center])
                code |= (1 << n);
        }
    }

    output_codes[center] = code;

    int* number = &(histogram[code]);
    atomicAdd(number, 1);
}

int main()
{
    Mat image = imread("C:/sobel.jpg", IMREAD_GRAYSCALE);

    int radius_neighbors = 1;
    int count_neighbors = 8; 

    int blocks_x = (image.cols + 31) / 32; 
    int blocks_y = (image.rows + 31) / 32;

    uchar* d_input;
    hipMalloc(&d_input, image.total() * sizeof(uchar));
    hipMemcpy(d_input, image.data, image.total() * sizeof(uchar), hipMemcpyHostToDevice);

    int* d_output;
    hipMalloc(&d_output, image.total() * sizeof(int));

    int size_histogram = 256;
    int* d_histogram;
    hipMalloc(&d_histogram, size_histogram * sizeof(int));

    calculate_lbp <<<dim3(blocks_x, blocks_y), dim3(32, 32)>>>(d_input, d_output, d_histogram, image.cols, image.rows, radius_neighbors, count_neighbors);

    int* h_output = new int[image.total()];
    int* h_histogram = new int[size_histogram];

    hipMemcpy(h_output, d_output, image.total() * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_histogram, d_histogram, size_histogram * sizeof(int), hipMemcpyDeviceToHost);

    Mat output_image = Mat(image.size(), CV_8UC1);
    for (int y = 0; y < image.rows; y++)
    {
        uchar* pixels = output_image.ptr<uchar>(y);
        for (int x = 0; x < image.cols; x++)
            pixels[x] = h_output[y * image.cols + x];
    }

    hipFree(d_input);
    hipFree(d_output);

    delete[] h_histogram;
    delete[] h_output;

    cv::imshow("image", output_image);
    cv::waitKey(0);

    return 0;
}