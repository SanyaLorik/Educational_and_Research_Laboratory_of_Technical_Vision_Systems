#include "hip/hip_runtime.h"
#include <opencv2/core.hpp>
#include <opencv2/imgproc.hpp>
#include <opencv2/highgui.hpp>

#include <hip/hip_runtime.h>
#include <>
#include <> 
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

#include <iostream>

using namespace std;
using namespace cv;

__global__ void calculate_lbp(const uchar* input_image, int* output_codes, int* histogram, int width_image, int height_image, int radius_neighbors, int count_neighbors)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x; // �������� �������
    int y = blockIdx.y * blockDim.y + threadIdx.y; // �������� �������

    if (x >= width_image || y >= height_image)
        return;

    double const M_PI = 3.1415926535f;

    int center = y * width_image + x;

    unsigned char code = 0;
    for (int n = 0; n < count_neighbors; n++)
    {
        double angle = 2 * M_PI * n / count_neighbors;
        int neighbor_x = x + radius_neighbors * nearbyint(cos(angle));
        int neighbor_y = y + radius_neighbors * nearbyint(sin(angle));

        if (neighbor_x >= 0 && neighbor_x < width_image && neighbor_y >= 0 && neighbor_y < height_image)
        {
            if (input_image[neighbor_y * width_image + neighbor_x] >= input_image[center])
                code |= (1 << n);
        }
    }

    output_codes[center] = code;

    int* number = &(histogram[code]);
    atomicAdd(number, 1);
}

__global__ void gauss(const uchar* input_image, uchar* output_iamge, const float** kernel, int kernelSize, int width_image, int height_image)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width_image || y >= height_image)
        return;

    int centre = y * width_image + x;
    int sumColor = 0;

    for (int kx = 0; kx < kernelSize; kx++)
    {
        for (int ky = 0; ky < kernelSize; ky++)
        {
            if ((x + kx) >= 0 && (x + kx) < width_image && (y + ky) >= 0 && (y + ky) < height_image)
            {
                int offset = (y + ky) * width_image + (x + kx);

                float maskValue = kernel[ky][kx];
                sumColor += input_image[offset] * maskValue;
            }
        }
    }

    output_iamge[centre] = sumColor;
}

int main()
{
    Mat image = imread("C:/sobel.jpg", IMREAD_GRAYSCALE);

    int radius_neighbors = 1;
    int count_neighbors = 8;

    int blocks_x = (image.cols + 31) / 32;
    int blocks_y = (image.rows + 31) / 32;

    uchar* d_input_image;
    hipMalloc(&d_input_image, image.total() * sizeof(uchar));
    hipMemcpy(d_input_image, image.data, image.total() * sizeof(uchar), hipMemcpyHostToDevice);

    int* d_output_code;
    hipMalloc(&d_output_code, image.total() * sizeof(int));

    int size_histogram = 256;
    int* d_histogram;
    hipMalloc(&d_histogram, size_histogram * sizeof(int));

    calculate_lbp << <dim3(blocks_x, blocks_y), dim3(32, 32) >> > (
        d_input_image, d_output_code, d_histogram, 
        image.cols, image.rows, radius_neighbors, count_neighbors);

    int* h_output_code = new int[image.total()];
    int* h_histogram = new int[size_histogram];

    hipMemcpy(h_output_code, d_output_code, image.total() * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_histogram, d_histogram, size_histogram * sizeof(int), hipMemcpyDeviceToHost);

    Mat output_image = Mat(image.size(), CV_8UC1);
    for (int y = 0; y < image.rows; y++)
    {
        uchar* pixels = output_image.ptr<uchar>(y);
        for (int x = 0; x < image.cols; x++)
            pixels[x] = h_output_code[y * image.cols + x];
    }

    hipFree(d_input_image);
    hipFree(d_output_code);

    delete[] h_histogram;
    delete[] h_output_code;

    cv::imshow("image", output_image);
    cv::waitKey(0);

    return 0;
}