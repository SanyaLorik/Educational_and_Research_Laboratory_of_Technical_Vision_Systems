#include "hip/hip_runtime.h"
﻿#include <opencv2/core.hpp>
#include <opencv2/imgproc.hpp>
#include <opencv2/highgui.hpp>

#include <hip/hip_runtime.h>
#include <>
#include <> 
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

#include <iostream>

using namespace std;
using namespace cv;

/*
__global__ void gauss(
    const uchar* input_image, uchar* output_image, float* kernel, 
    int kernelSize, int width_image, int height_image)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width_image || y >= height_image)
        return;

    int centre = y * width_image + x;
    float sumColor = 0;

    for (int kx = 0; kx < kernelSize; kx++)
    {
        for (int ky = 0; ky < kernelSize; ky++)
        {
            if ((x + kx) >= 0 && (x + kx) < width_image && (y + ky) >= 0 && (y + ky) < height_image)
            {
                int offset = (y + ky) * width_image + (x + kx);

                float maskValue = kernel[ky * kernelSize + kx];
                sumColor += input_image[offset] * maskValue;
            }
        }
    }

    output_image[centre] = (uchar)sumColor;
}
*/

__global__ void gauss(
    const hipTextureObject_t input_image, uchar* output_image, float* kernel,
    int kernelSize, int width_image, int height_image)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    float sumColor = 0;

    for (int kx = 0; kx < kernelSize; kx++)
    {
        for (int ky = 0; ky < kernelSize; ky++)
        {
            float maskValue = kernel[ky * kernelSize + kx];
            sumColor += tex2D<uchar>(input_image, x + kx, y + ky) * maskValue;
        }
    }

    int centre = y * width_image + x;
    output_image[centre] = (uchar)sumColor;
}

float* createGaussianKernel(int kernelSize, double sigma);

float guassian(int x, int y, double sigma);

int main()
{
    Mat image = imread("C:/sobel.jpg", IMREAD_GRAYSCALE);

    int kernelSize = 11;
    double sigma = 8;
    float* kernel = createGaussianKernel(kernelSize, sigma);

    int block_y = (image.rows + 31) / 32;
    int block_x = (image.cols + 31) / 32;

    uchar* d_input_image;
    hipMalloc(&d_input_image, image.total() * sizeof(uchar));
    hipMemcpy(d_input_image, image.data, image.total() * sizeof(uchar), hipMemcpyHostToDevice);

    uchar* d_output_image;
    hipMalloc(&d_output_image, image.total() * sizeof(uchar));

    float* d_kernel;
    hipMalloc(&d_kernel, kernelSize * kernelSize * sizeof(float));
    hipMemcpy(d_kernel, kernel, kernelSize * kernelSize * sizeof(float), hipMemcpyHostToDevice);

    hipTextureObject_t texObj;

    hipArray* d_array;
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<uchar>();
    hipMallocArray(&d_array, &channelDesc, image.cols, image.rows);
    
    hipMemcpyToArray(d_array, 0, 0, image.data, image.total() * sizeof(uchar), hipMemcpyHostToDevice);

    hipResourceDesc resDesc = {};
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = d_array;

    hipTextureDesc texDesc = {};
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModePoint;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 0;

    hipCreateTextureObject(&texObj, &resDesc, &texDesc, nullptr);

    gauss <<<dim3(block_x, block_y), dim3(32, 32)>>> (
        texObj, d_output_image, d_kernel,
        kernelSize, image.cols, image.rows);

    uchar* h_output_image = new uchar[image.total()];

    hipMemcpy(h_output_image, d_output_image, image.total() * sizeof(uchar), hipMemcpyDeviceToHost);
    
    Mat output_image = Mat(image.size(), CV_8UC1);
    for (int y = 0; y < image.rows; y++)
    {
        uchar* pixels = output_image.ptr<uchar>(y);
        for (int x = 0; x < image.cols; x++)
            pixels[x] = h_output_image[y * image.cols + x];
    }

    hipFree(d_input_image);
    hipFree(d_output_image);
    hipFree(d_kernel);

    delete[] h_output_image;

    cv::imshow("image", output_image);
    cv::waitKey(0);

    return 0;
}

float* createGaussianKernel(int kernelSize, double sigma) 
{
    float* kernel = new float[kernelSize * kernelSize];

    int indent = kernelSize / 2;
    float sum = 0.0f;

    for (int y = -indent; y <= indent; y++)
    {
        for (int x = -indent; x <= indent; x++) 
        {
            float value = guassian(x, y, sigma);
            sum += value;
            kernel[(y + indent) * kernelSize + (x + indent)] = value;
        }
    }

    for (int i = 0; i < kernelSize * kernelSize; i++) 
        kernel[i] /= sum;

    return kernel;
}

float guassian(int x, int y, double sigma)
{
    return (1 / (2 * 3.1415f * powf(sigma, 2))) * expf(-((x * x + y * y) / (2 * powf(sigma, 2))));
}